#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h> // needed for timing
#include "derivative.h"
#include "Fields.h"
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_cooperative_groups.h>

#define NX_VALUE 110
#define NY_VALUE 110
namespace cg = cooperative_groups;

typedef double gpu_fp;

double get_time(void);
void initFluid(struct Fluid *fluid, int Nx, int Ny,int ghost_zone,double dx,double dy);

/*
 * CFLAGS=-lm -I.
DEPS = derivative.h Fields.h
OBJ = main.o Fields.o

%.o: %.c $(DEPS)
	gcc -Wall -fopenmp -O3 --fast-math -c -o $@ $< $(CFLAGS)

Simul: $(OBJ)
	gcc -Wall -fopenmp -O3 --fast-math -o $@ $^ $(CFLAGS)
clean:
	-rm Simul:
 */
struct Fluid{
    double density;
    double viscosity;
    struct Field2D* speed;
    struct Field2D* old;
    struct FieldScalar* pressure;
    struct Field2D* dot;
    struct Field2D* temp2D;
    struct FieldScalar* tempScalar;
    struct FieldScalar* smoke;
    char* mask;
};
void output_gf(struct Fluid* fluid);
/*
__device__
void forces(struct Field2D* dot,double t,int i,int j){
    const int Nx=dot->Nx;
    const int Ny=dot->Ny;
    const double dx=dot->dx;
    const double dy=dot->dy;

    int index=INDEX_GPU(i,j,dot->Nx);
    dot->cx[index]=0;
    dot->cy[index]=0;


}
__device__
void derivative_no_pressure(struct Fluid* fluid,double time_t,int i,int j){
    const int Nx=fluid->speed->Nx;
    const int Ny=fluid->speed->Ny;
    const double dx=fluid->speed->dx;
    const double dy=fluid->speed->dy;
    const int ghost_zone=fluid->speed->ghost_zone;
    forces(fluid->dot,time_t,i,j);

    double viscOverDens=fluid->viscosity/fluid->density;
    int index=INDEX_GPU(i,j,Nx);
    double term_laplacien=viscOverDens*(SD_2O_CEN_GPU(fluid->speed->cx,index,1,dx)+
            SD_2O_CEN_GPU(fluid->speed->cy,index,Nx,dy));
    fluid->dot->cx[index]+= term_laplacien;
    fluid->dot->cy[index]+= term_laplacien;
}
__device__
void set_boundaries(struct Fluid* fluid,double time_t,int i,int j){
    const int Nx=fluid->speed->Nx;
    const int Ny=fluid->speed->Ny;
    const double dx=fluid->speed->dx;
    const double dy=fluid->speed->dy;

    //y=0 boundary
    if(j==0) {
        fluid->speed->cx[i] = 0;
        fluid->speed->cy[i] = 0;
    }
    //y=1 boundary
    if(j==Ny-1) {
        fluid->speed->cx[i + Nx * (Ny - 1)] = 0;
        fluid->speed->cy[i + Nx * (Ny - 1)] = 0;
    }

    if(i==0) {
        fluid->speed->cx[Nx*j]=0;
        fluid->speed->cy[Nx*j]=0;
        fluid->smoke->v[Nx*j]=0;
        if(j>80+40* cos(0.25*time_t) && j<100+40* cos(0.25*time_t)){
            fluid->speed->cx[Nx*j]=12;
            fluid->smoke->v[Nx*j]=1;
        }
    }
    if(i==Nx-1){
        fluid->speed->cx[Nx*j+Nx-1]=12*20.0/Ny;
        fluid->speed->cy[Nx*j+Nx-1]=0;
        fluid->smoke->v[Nx*j+Nx-1]=0;
    }


    int index = INDEX_GPU(i, j, Nx);
    if(fluid->mask[index]==0){
        fluid->speed->cx[index]=0;
        fluid->speed->cy[index]=0;
        fluid->smoke->v[index]=-1;
    }

}

__global__
void RK4(struct Fluid* fluid,double time_t,double dt){
    //__shared__ double shared_v[NY_VALUE];
    //__shared__ double shared_cx[NY_VALUE];
    //__shared__ double shared_cy[NY_VALUE];

    int j =  threadIdx.x;
    int i = blockIdx.x;

    const int Nx=fluid->speed->Nx;
    const int Ny=fluid->speed->Ny;
    const double dx=fluid->speed->dx;
    const double dy=fluid->speed->dy;
    const int ghost_zone=fluid->speed->ghost_zone;

    int index = INDEX_GPU(i, j, Nx);
    double hdt = 0.5 * dt;
    double dt_6 = dt / 6;
    double dt_3 = dt / 3;

    if (i >= 0 && j >=0 && i < Nx && j < Ny){
        if (fluid->mask[index] != 0 && i >= ghost_zone && j >=ghost_zone && i < Nx-ghost_zone && j < Ny-ghost_zone)derivative_no_pressure(fluid, time_t,i,j);
        else set_boundaries(fluid, time_t,i,j);
    }

    __syncthreads();
    if (i >= ghost_zone && j >=ghost_zone && i < Nx-ghost_zone && j < Ny-ghost_zone)
        if (fluid->mask[index] != 0) {
            fluid->old->cx[index] = fluid->speed->cx[index];
            fluid->old->cy[index] = fluid->speed->cy[index];

            fluid->temp2D->cx[index] = fluid->speed->cx[index] + dt_6 * fluid->dot->cx[index];
            fluid->temp2D->cy[index] = fluid->speed->cy[index] + dt_6 * fluid->dot->cy[index];

            fluid->speed->cx[index] += hdt * fluid->dot->cx[index];
            fluid->speed->cy[index] += hdt * fluid->dot->cy[index];
        }
    __syncthreads();
    if (i >= 0 && j >=0 && i < Nx && j < Ny){
        if (fluid->mask[index] != 0 && i >= ghost_zone && j >=ghost_zone && i < Nx-ghost_zone && j < Ny-ghost_zone)derivative_no_pressure(fluid, time_t,i,j);
        else set_boundaries(fluid, time_t,i,j);
    }
    __syncthreads();
    if (i >= ghost_zone && j >=ghost_zone && i < Nx-ghost_zone && j < Ny-ghost_zone)
        if (fluid->mask[index] != 0) {
            fluid->temp2D->cx[index] += dt_3 * fluid->dot->cx[index];
            fluid->temp2D->cy[index] += dt_3 * fluid->dot->cy[index];

            fluid->speed->cx[index] += fluid->old->cx[index] + hdt * fluid->dot->cx[index];
            fluid->speed->cy[index] += fluid->old->cy[index] + hdt * fluid->dot->cy[index];
        }
    __syncthreads();
    if (i >= 0 && j >=0 && i < Nx && j < Ny){
        if (fluid->mask[index] != 0 && i >= ghost_zone && j >=ghost_zone && i < Nx-ghost_zone && j < Ny-ghost_zone)derivative_no_pressure(fluid, time_t,i,j);
        else set_boundaries(fluid, time_t,i,j);
    }

    if (i >= ghost_zone && j >=ghost_zone && i < Nx-ghost_zone && j < Ny-ghost_zone)
        if (fluid->mask[index] != 0) {
            fluid->temp2D->cx[index] += dt_3 * fluid->dot->cx[index];
            fluid->temp2D->cy[index] += dt_3 * fluid->dot->cy[index];

            fluid->speed->cx[index] += fluid->old->cx[index] + dt * fluid->dot->cx[index];
            fluid->speed->cy[index] += fluid->old->cy[index] + dt * fluid->dot->cy[index];
        }
    __syncthreads();
    if (i >= 0 && j >=0 && i < Nx && j < Ny){
        if (fluid->mask[index] != 0 && i >= ghost_zone && j >=ghost_zone && i < Nx-ghost_zone && j < Ny-ghost_zone)derivative_no_pressure(fluid, time_t,i,j);
        else set_boundaries(fluid, time_t,i,j);
    }
    __syncthreads();
    if (i >= ghost_zone && j >=ghost_zone && i < Nx-ghost_zone && j < Ny-ghost_zone)
        if (fluid->mask[index] != 0) {
            fluid->speed->cx[index]=fluid->temp2D->cx[index]+dt_6* fluid->dot->cx[index];
            fluid->speed->cy[index]=fluid->temp2D->cy[index]+dt_6* fluid->dot->cy[index];
        }

}
*/
void forces(struct Field2D* dot,double t){
    for (int j = dot->ghost_zone; j < dot->Ny-dot->ghost_zone; ++j) {
        for (int i = dot->ghost_zone; i < dot->Nx-dot->ghost_zone; ++i) {
            int index=INDEX(i,j,dot->Nx);
            dot->cx[index]=0;
            dot->cy[index]=0;
            /* double espx=i-50;
              if(espx>0){
                  espx= fmax(0,espx-25);
              }
              espx=(espx*espx)*dot->dx*dot->dx;
              double espy=(j-30.5)*(j-30.5)*dot->dy*dot->dy;
              dot->cx[index]=  fmax(-6,-3/sqrt(espx+espy));

              espy=(j-30.5)*(j-80.5)*dot->dy*dot->dy;
              dot->cx[index]= fmax(-6,-3/sqrt(espx+espy));*/

        }
    }
}

void derivative_no_pressure(struct Fluid* fluid,double time_t){
    const int Nx=fluid->speed->Nx;
    const int Ny=fluid->speed->Ny;
    const double dx=fluid->speed->dx;
    const double dy=fluid->speed->dy;
    const int ghost_zone=fluid->speed->ghost_zone;
    forces(fluid->dot,time_t);

    double viscOverDens=fluid->viscosity/fluid->density;
    double max1=0;
    for (int j = ghost_zone; j < Ny-ghost_zone; ++j) {
        for (int i = ghost_zone; i < Nx-ghost_zone; ++i) {
            int index=INDEX(i,j,Nx);
            double term_laplacien=viscOverDens*(SD_2O_CEN(fluid->speed->cx,index,1,dx)+
                                                SD_2O_CEN(fluid->speed->cy,index,Nx,dy));
            max1=fmax(max1,fabs(term_laplacien));
            fluid->dot->cx[index]+= term_laplacien;
            fluid->dot->cy[index]+= term_laplacien;
            /* fluid->dot->cx[index]+= fluid->speed->cx[index]* FD_2O_CEN(fluid->speed->cx,index,1,dx)+
                     fluid->speed->cy[index]* FD_2O_CEN(fluid->speed->cy,index,1,dx);
             fluid->dot->cy[index]+= fluid->speed->cx[index]* FD_2O_CEN(fluid->speed->cx,index,Nx,dy)+
                                     fluid->speed->cy[index]* FD_2O_CEN(fluid->speed->cy,index,Nx,dy);
          */
        }
    }

}

void set_boundaries(struct Fluid* fluid,double time_t){
    const int Nx=fluid->speed->Nx;
    const int Ny=fluid->speed->Ny;
    const double dx=fluid->speed->dx;
    const double dy=fluid->speed->dy;
    const int ghost_zone=fluid->speed->ghost_zone;
    for (int i = 0; i < Nx; ++i) {
        //y=0 boundary
        fluid->speed->cx[i]=0;
        fluid->speed->cy[i]=0;
        //y=1 boundary
        fluid->speed->cx[i+Nx*(Ny-1)]=0;
        fluid->speed->cy[i+Nx*(Ny-1)]=0;
    }
    for (int j = 0; j < Ny; ++j) {
        //x=0 boundary
        fluid->speed->cx[Nx*j]=0;

        fluid->speed->cy[Nx*j]=0;
        //x=1 boundary
        fluid->speed->cx[Nx*j+Nx-1]=12*20.0/Ny;
        fluid->speed->cy[Nx*j+Nx-1]=0;
        fluid->smoke->v[Nx*j]=0;
        fluid->smoke->v[Nx*j+Nx-1]=0;
        if(j>40+20* cos(0.25*time_t) && j<60+20* cos(0.25*time_t)){
            fluid->speed->cx[Nx*j]=12;
            fluid->smoke->v[Nx*j]=1;
            //printf("%f %f\n",40+20* cos(0.5*time_t),time_t);
        }
    }
    for (int j = ghost_zone; j < Ny-ghost_zone; ++j) {
        for (int i = ghost_zone; i < Nx - ghost_zone; ++i) {
            int index = INDEX(i, j, Nx);
            if(fluid->mask[index]>0)continue;
            fluid->speed->cx[index]=0;
            fluid->speed->cy[index]=0;
            fluid->smoke->v[index]=-1;
        }
    }
}


void RK4(struct Fluid* fluid,double time_t,double dt){
    const int Nx=fluid->speed->Nx;
    const int Ny=fluid->speed->Ny;
    const int ghost_zone=fluid->speed->ghost_zone;

    double hdt=0.5*dt;
    double dt_6=dt/6;
    double dt_3=dt/3;
    derivative_no_pressure(fluid,time_t);
    set_boundaries(fluid,time_t);
    double vmax=0;
    for (int j = ghost_zone; j < Ny-ghost_zone; ++j){
        for (int i = ghost_zone; i < Nx-ghost_zone; ++i) {
            int index=INDEX(i,j,Nx);
            if(fluid->mask[index]==0)continue;
            fluid->old->cx[index]=fluid->speed->cx[index];
            fluid->old->cy[index]=fluid->speed->cy[index];

            fluid->temp2D->cx[index]=fluid->speed->cx[index]+dt_6* fluid->dot->cx[index];
            fluid->temp2D->cy[index]=fluid->speed->cy[index]+dt_6* fluid->dot->cy[index];

            fluid->speed->cx[index]+=hdt* fluid->dot->cx[index];
            fluid->speed->cy[index]+=hdt* fluid->dot->cy[index];
            vmax= fmax(fmax(fabs(fluid->speed->cx[index]),fabs(fluid->speed->cy[index])),vmax);
        }
    }

    vmax=0;
    derivative_no_pressure(fluid,time_t+hdt);
    set_boundaries(fluid,time_t+hdt);
    for (int j = ghost_zone; j < Ny-ghost_zone; ++j){
        for (int i = ghost_zone; i < Nx-ghost_zone; ++i) {
            int index=INDEX(i,j,Nx);
            if(fluid->mask[index]==0)continue;
            fluid->temp2D->cx[index]+=dt_3* fluid->dot->cx[index];
            fluid->temp2D->cy[index]+=dt_3* fluid->dot->cy[index];

            fluid->speed->cx[index]+=fluid->old->cx[index]+hdt* fluid->dot->cx[index];
            fluid->speed->cy[index]+=fluid->old->cy[index]+hdt* fluid->dot->cy[index];
        }
    }
    derivative_no_pressure(fluid,time_t+hdt);
    set_boundaries(fluid,time_t+hdt);

    for (int j = ghost_zone; j < Ny-ghost_zone; ++j){
        for (int i = ghost_zone; i < Nx-ghost_zone; ++i) {
            int index=INDEX(i,j,Nx);
            if(fluid->mask[index]==0)continue;
            fluid->temp2D->cx[index]+=dt_3* fluid->dot->cx[index];
            fluid->temp2D->cy[index]+=dt_3* fluid->dot->cy[index];

            fluid->speed->cx[index]+=fluid->old->cx[index]+dt* fluid->dot->cx[index];
            fluid->speed->cy[index]+=fluid->old->cy[index]+dt* fluid->dot->cy[index];
        }
    }

    derivative_no_pressure(fluid,time_t+dt);
    set_boundaries(fluid,time_t+dt);
    for (int j = ghost_zone; j < Ny-ghost_zone; ++j){
        for (int i = ghost_zone; i < Nx-ghost_zone; ++i) {
            int index=INDEX(i,j,Nx);
            if(fluid->mask[index]==0)continue;
            fluid->speed->cx[index]=fluid->temp2D->cx[index]+dt_6* fluid->dot->cx[index];
            fluid->speed->cy[index]=fluid->temp2D->cy[index]+dt_6* fluid->dot->cy[index];
            vmax= fmax(fmax(fabs(fluid->speed->cx[index]),fabs(fluid->speed->cy[index])),vmax);
        }
    }
}
void solve_pressure2(struct Fluid* fluid,double time_t,double dt){
    const int Nx=fluid->speed->Nx;
    const int Ny=fluid->speed->Ny;
    const double dx=fluid->speed->dx;
    const double dy=fluid->speed->dy;
    const int ghost_zone=fluid->speed->ghost_zone;
    divergence(fluid->speed,fluid->tempScalar);
    for (int ite = 0; ite < 10000; ++ite) {
        double error=0;
        double maxe=0;

        divergence(fluid->speed,fluid->tempScalar);
        for (int j = ghost_zone; j < Ny-ghost_zone; ++j) {
            for (int i = ghost_zone; i < Nx-ghost_zone; ++i) {
                int index=INDEX(i,j,Nx);
                if(fluid->mask[index]==0)continue;
                double value_e=fluid->tempScalar->v[index];
                error+=value_e*value_e;
                maxe= fmax(fabs(value_e),maxe);
            }
        }
        error/=(Nx-2)*(Ny-2);
        if(error<1e-5 ||ite>=1000){
            printf("Iteration: %d;Error:%20.16e Max:%20.16e\n",ite,error,maxe);
            break;
        }

        if(ite==0){
            printf("Initialisation Error:%20.16e Max:%20.16e\n",error,maxe);
        }

        for (int k = 0; k < 100; ++k) {
            for (int j = ghost_zone; j < Ny-ghost_zone; ++j) {
                for (int i = ghost_zone; i < Nx-ghost_zone; ++i) {
                    int index=INDEX(i,j,Nx);
                    if(fluid->mask[index]==0)continue;
                    double div=2*dx*FD_2O_CEN(fluid->speed->cx,INDEX(i,j,Nx),1,dx)+2*dy*FD_2O_CEN(fluid->speed->cy,INDEX(i,j,Nx),Nx,dy);
                    int count_neighs=4;
                    if(fluid->mask[INDEX(i-1,j,Nx)]==0)count_neighs--;
                    if(fluid->mask[INDEX(i+1,j,Nx)]==0)count_neighs--;
                    if(fluid->mask[INDEX(i,j-1,Nx)]==0)count_neighs--;
                    if(fluid->mask[INDEX(i,j+1,Nx)]==0)count_neighs--;

                    fluid->speed->cy[INDEX(i,j-1,Nx)]+=div/count_neighs*1.5;
                    fluid->speed->cy[INDEX(i,j+1,Nx)]-=div/count_neighs*1.5;
                    fluid->speed->cx[INDEX(i-1,j,Nx)]+=div/count_neighs*1.5;
                    fluid->speed->cx[INDEX(i+1,j,Nx)]-=div/count_neighs*1.5;
                    fluid->pressure->v[index]+=div/4*fluid->density*dx/dt;
                }
            }
            //set_boundaries(fluid,time_t);
        }
    }

}

__global__
void advect(struct Fluid* fluid,double delta_t){
    __shared__ double shared_v[NY_VALUE];
    __shared__ double shared_cx[NY_VALUE];
    __shared__ double shared_cy[NY_VALUE];

    int j =  threadIdx.x;
    int i_stride=gridDim.x;
    int i = blockIdx.x;
    //cg::grid_group grid = cg::this_grid();
    const int Nx=fluid->speed->Nx;
    const int Ny=fluid->speed->Ny;
    const double dx=fluid->speed->dx;
    const double dy=fluid->speed->dy;
    const int ghost_zone=fluid->speed->ghost_zone;

    int index = INDEX_GPU(i, j, Nx);


    if (i >= ghost_zone && j >= ghost_zone && i < Nx - ghost_zone && j < Ny - ghost_zone)
        if (fluid->mask[index] != 0) {
            double newX = i - delta_t / dx * fluid->speed->cx[index];
            double newY = j - delta_t / dy * fluid->speed->cy[index];
            newX = fmax(0.0, fmin(0.0 + Nx - 1, newX));
            newY = fmax(0.0, fmin(0.0 + Ny - 1, newY));
            int nx = (int) newX;
            int ny = (int) newY;

            double difx = newX - nx;
            double dify = newY - ny;
            shared_cx[index] = (1 - difx) * (1 - dify) * fluid->speed->cx[INDEX_GPU(nx, ny, Nx)]
                                       + (difx) * (1 - dify) * fluid->speed->cx[INDEX_GPU(nx + 1, ny, Nx)]
                                       + (1 - difx) * (dify) * fluid->speed->cx[INDEX_GPU(nx, ny + 1, Nx)]
                                       + (difx) * (dify) * fluid->speed->cx[INDEX_GPU(nx + 1, ny + 1, Nx)];
            shared_cy[index] = (1 - difx) * (1 - dify) * fluid->speed->cy[INDEX_GPU(nx, ny, Nx)]
                                       + (difx) * (1 - dify) * fluid->speed->cy[INDEX_GPU(nx + 1, ny, Nx)]
                                       + (1 - difx) * (dify) * fluid->speed->cy[INDEX_GPU(nx, ny + 1, Nx)]
                                       + (difx) * (dify) * fluid->speed->cy[INDEX_GPU(nx + 1, ny + 1, Nx)];
            shared_v[index] =
                    (1 - difx) * (1 - dify) * fmax(fluid->smoke->v[INDEX_GPU(nx, ny, Nx)], 0.0)
                    + (difx) * (1 - dify) * fmax(fluid->smoke->v[INDEX_GPU(nx + 1, ny, Nx)], 0.0)
                    + (1 - difx) * (dify) * fmax(fluid->smoke->v[INDEX_GPU(nx, ny + 1, Nx)], 0.0)
                    + (difx) * (dify) * fmax(fluid->smoke->v[INDEX_GPU(nx + 1, ny + 1, Nx)], 0.0);

        }

    __syncthreads();

    if (i >= ghost_zone && j >= ghost_zone && i < Nx - ghost_zone && j < Ny - ghost_zone)
        if (fluid->mask[index] != 0) {
            shared_cx[index] = fluid->temp2D->cx[index];
            shared_cy[index] = fluid->temp2D->cy[index];
            shared_v[index] = fluid->tempScalar->v[index];
        }

}


int main(int argc, char **argv) {
    const int Nx=NX_VALUE;
    const int Ny=NY_VALUE;
    const int ghost_zone=1;
    const double dx=70.0/(Nx-1);
    const double dy=70.0/(Ny-1);
    double dt=0.05;
    struct Fluid* fluid;
    CUDA_CHECK(hipMallocManaged(&fluid, sizeof(struct Fluid)));
    printf("Entraine\n");
    initFluid(fluid,Nx,Ny,ghost_zone,dx,dy);
    printf("Fin init\n");

    char* centered_mask;
    CUDA_CHECK(hipMallocManaged(&(centered_mask), Nx * Ny * sizeof(char)));
    FILE *fichier;
    // Chemin du fichier à ouvrir en mode lecture ("r" pour read)
    const char *chemin_fichier = "duck110.txt";
    fichier = fopen(chemin_fichier, "r");
    // Vérifier si l'ouverture du fichier a réussi
    if (fichier == NULL) {
        fprintf(stderr, "Erreur lors de l'ouverture du fichier.\n");
        return 1; // Quitter le programme avec code d'erreur
    }
    double value=0;
    int count=0;
    while (fscanf(fichier, "%lf", &value) == 1) {
        if(count>=Nx*Ny)break;
        int i=count/Nx;
        int j=count%Nx;

        centered_mask[count]=1;
        fluid->mask[count]=1;
        if(value<0.5 && i>5 && i<Nx-5 && j>5 && j<Ny-5){
            centered_mask[count]=0;
            fluid->mask[count]=0;
            fluid->smoke->v[count]=-1;
        }
        count++;
    }
    printf("Fin %d/%d\n",count,Nx*Ny);
    fclose(fichier);

    fluid->density=1;
    fluid->viscosity=0.001;


    int numBlocksPerSm = 0;
    // Number of threads my_kernel will be launched with
    int numThreads = Ny;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksPerSm, (void*)advect, numThreads, 0);
    // launch
    void *kernelArgs[] = { fluid };
    dim3 dimBlock(numThreads, 1, 1);
    dim3 dimGrid(deviceProp.multiProcessorCount*numBlocksPerSm, 1, 1);
    /*if (dimGrid.x > Nx)
    {
        dimGrid.x = Nx;
    }*/
    printf("numBlocks = %d, blockSize = %d, totalThreads=%d\n", dimGrid.x, dimBlock.x, dimGrid.x * dimBlock.x);

    double  begin=get_time();
    output_gf(fluid);
    double t=0;
    int iterations=0;
    while(t<=10){
        printf("Time:%f\n",t);
        RK4(fluid,t,dt);//<<<Nx, Ny>>>
        //CUDA_CHECK(hipDeviceSynchronize());
        solve_pressure2(fluid,t,dt);

        advect <<<Nx, Ny>>> (fluid,dt);
        CUDA_CHECK(hipDeviceSynchronize());
        t+=dt;
        iterations+=1;
        if(iterations%5==0){
            output_gf(fluid);
        }

    }

    printf("Fin time:%f Iteration:%d\n",get_time()-begin,iterations);

    freeFieldScalar(fluid->pressure);
    freeFieldScalar(fluid->tempScalar);
    freeField3D(fluid->speed);
    freeField3D(fluid->dot);
    freeField3D(fluid->temp2D);
    freeField3D(fluid->old);
    freeFieldScalar(fluid->smoke);
    hipFree(centered_mask);
    hipFree(fluid->speed);
    hipFree(fluid->dot);
    hipFree(fluid->temp2D);
    hipFree(fluid->tempScalar);
    hipFree(fluid->pressure);
    hipFree(fluid->old);
    hipFree(fluid->smoke);
    hipFree(fluid->mask);
    return 0;
}



void initFluid(struct Fluid *fluid, int Nx, int Ny,int ghost_zone,double dx,double dy) {

    // Allouer de la mémoire pour chaque structure et tableau sur le GPU
    CUDA_CHECK(hipMallocManaged(&(fluid->speed), sizeof(struct Field2D)));
    CUDA_CHECK(hipMallocManaged(&(fluid->dot), sizeof(struct Field2D)));
    CUDA_CHECK(hipMallocManaged(&(fluid->temp2D), sizeof(struct Field2D)));
    CUDA_CHECK(hipMallocManaged(&(fluid->old), sizeof(struct Field2D)));
    CUDA_CHECK(hipMallocManaged(&(fluid->pressure), sizeof(struct FieldScalar)));
    CUDA_CHECK(hipMallocManaged(&(fluid->tempScalar), sizeof(struct FieldScalar)));
    CUDA_CHECK(hipMallocManaged(&(fluid->smoke), sizeof(struct FieldScalar)));
    CUDA_CHECK(hipMallocManaged(&(fluid->mask), Nx * Ny * sizeof(char)));

    if (fluid->speed == NULL || fluid->dot == NULL || fluid->temp2D == NULL ||
        fluid->old == NULL || fluid->pressure == NULL || fluid->tempScalar == NULL ||
        fluid->smoke == NULL || fluid->mask == NULL) {
        printf("Error dynamic allocation for fluid\n");
        exit(-1);
    }
    // Initialiser d'autres champs si nécessaire
    initField2D(fluid->speed,Nx,Ny,ghost_zone,dx,dy);
    initField2D(fluid->dot,Nx,Ny,ghost_zone,dx,dy);
    initField2D(fluid->temp2D,Nx,Ny,ghost_zone,dx,dy);
    initField2D(fluid->old,Nx,Ny,ghost_zone,dx,dy);
    initFieldScalar(fluid->pressure,Nx,Ny,ghost_zone,dx,dy);
    initFieldScalar(fluid->tempScalar,Nx,Ny,ghost_zone,dx,dy);
    initFieldScalar(fluid->smoke,Nx,Ny,ghost_zone,dx,dy);
}

void output_gf(struct Fluid* fluid)
{

    static int counter = 0;
    const int Nx=fluid->speed->Nx;
    const int Ny=fluid->speed->Ny;
    const double dx=fluid->speed->dx;
    const double dy=fluid->speed->dy;

    char name_buff[1024];
    snprintf(name_buff, 1024, "results/%07d.asc", counter);
    FILE *ofile = fopen(name_buff, "w");
    for (int j = 0; j < Ny; ++j)
    {
        for (int i = 0; i < Nx; ++i)
        {
            const int ij = INDEX(i, j, Nx);
            fprintf(ofile, "%20.16e %20.16e %20.16e %20.16e %20.16e %20.16e\n", i * dx,j * dy,
                    fluid->pressure->v[ij],fluid->speed->cx[ij],fluid->speed->cy[ij],fluid->smoke->v[ij]);
        }
    }
    ++counter;

    fclose(ofile);
    ofile = NULL;
}

double get_time(void)
{
    struct timeval tv;
    gettimeofday(&tv, NULL);

    return (tv.tv_sec) + 1.0e-6 * tv.tv_usec;
}



