#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h> // needed for timing
#include "derivative.h"
#include "Fields.h"
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_cooperative_groups.h>
#define NX_VALUE 150
#define NY_VALUE 150
#define USUAL_VX_OUTPUT 12
#define USUAL_VX_INPUT 12 //
namespace cg = cooperative_groups;

typedef double gpu_fp;

double get_time(void);
void initFluid(struct Fluid *fluid, int Nx, int Ny,int ghost_zone,double dx,double dy);

struct Fluid{
    double density;
    double viscosity;
    double t;
    double dt;
    struct Field2D* speed;
    struct Field2D* old;
    struct FieldScalar* pressure;
    struct Field2D* temp2D;
    struct FieldScalar* tempScalar;
    struct FieldScalar* smoke;
    char* mask;
};

struct Params{
    struct Fluid* fluid;
    double time_tp;
    double dtp;
};

void output_gf(struct Fluid* fluid);
//use __shared for dot

__device__
void forces(double* dot_cx,double* dot_cy,double t,int i,int j){
    /*const int Nx=dot->Nx;
    const int Ny=dot->Ny;
    const double dx=dot->dx;
    const double dy=dot->dy;
     int index=INDEX_GPU(i,j,dot->Nx);
     */


    dot_cx[j]=0;
    dot_cy[j]=0;
}
__device__
void derivative_no_pressure(struct Fluid* fluid,double* dot_cx,double* dot_cy,double time_t,int i,int j){
    const int Nx=fluid->speed->Nx;
    //const int Ny=fluid->speed->Ny;
    const double dx=fluid->speed->dx;
    const double dy=fluid->speed->dy;
    //const int ghost_zone=fluid->speed->ghost_zone;
    forces(dot_cx,dot_cy,time_t,i,j);

    double viscOverDens=fluid->viscosity/fluid->density;
    int index=INDEX_GPU(i,j,Nx);
    double term_laplacien=viscOverDens*(SD_2O_CEN_GPU(fluid->speed->cx,index,1,dx)+
                                        SD_2O_CEN_GPU(fluid->speed->cy,index,Nx,dy));
    dot_cx[j]+= term_laplacien;
    dot_cy[j]+= term_laplacien;
}
__device__
void set_boundaries(struct Fluid* fluid,double time_t,int i,int j){
    const int Nx=fluid->speed->Nx;
    const int Ny=fluid->speed->Ny;
    const double dx=fluid->speed->dx;
    const double dy=fluid->speed->dy;

    //y=0 boundary
    if(j==0) {
        fluid->speed->cx[i] = 0;
        fluid->speed->cy[i] = 0;
    }
    //y=1 boundary
    if(j==Ny-1) {
        fluid->speed->cx[i + Nx * (Ny - 1)] = 0;
        fluid->speed->cy[i + Nx * (Ny - 1)] = 0;
    }

    if(i==0) {
        fluid->speed->cx[Nx*j]=USUAL_VX_INPUT;
        if(j>55)fluid->speed->cx[Nx*j]=-0;
        fluid->speed->cy[Nx*j]=0;
        fluid->smoke->v[Nx*j]=0;
       /* if(j>40+20* cos(0.25*time_t) && j<60+20* cos(0.25*time_t)){
            fluid->speed->cx[Nx*j]=12;
            fluid->smoke->v[Nx*j]=1;
        }*/
        if(j>30 && j<40 ){
            fluid->smoke->v[Nx*j]=1;
        }
    }

    if(i==Nx-1){
        fluid->speed->cx[Nx*j+Nx-1]=0;
        if(j>50)fluid->speed->cx[Nx*j+Nx-1]=-USUAL_VX_OUTPUT;
        fluid->speed->cy[Nx*j+Nx-1]=0;
        fluid->smoke->v[Nx*j+Nx-1]=0;
        if(j>70 && j<80){
            fluid->smoke->v[Nx*j+Nx-1]=1;
        }
    }


    int index = INDEX_GPU(i, j, Nx);
    if(fluid->mask[index]==0){
        fluid->speed->cx[index]=0;
        fluid->speed->cy[index]=0;
        fluid->smoke->v[index]=-1;
    }

}


__global__
void RK4(struct Fluid* fluid,double time_t,double dt){

    cooperative_groups::grid_group g = cooperative_groups::this_grid();
    dt= fluid->dt;
    time_t=fluid->t;
    __shared__ double shared_cx[NY_VALUE];
    __shared__ double shared_cy[NY_VALUE];
    __shared__ double old_cx[NY_VALUE];
    __shared__ double old_cy[NY_VALUE];
    __shared__ double dot_cx[NY_VALUE];
    __shared__ double dot_cy[NY_VALUE];
    int j =  threadIdx.x;
    //int i = blockIdx.x;

    const int Nx=fluid->speed->Nx;
    const int Ny=fluid->speed->Ny;
    const double dx=fluid->speed->dx;
    const double dy=fluid->speed->dy;
    const int ghost_zone=fluid->speed->ghost_zone;

    double hdt = 0.5 * dt;
    double dt_6 = dt / 6;
    double dt_3 = dt / 3;

    int rep=(Nx+gridDim.x-1)/gridDim.x;
    for(int id=0;id<rep;id++) {
        int i = blockIdx.x + id * gridDim.x;
        int index = INDEX_GPU(i, j, Nx);
        int mask = i >= ghost_zone && j >= ghost_zone && i < Nx - ghost_zone &&
                   j < Ny - ghost_zone;
        if (mask)mask = fluid->mask[index] != 0;

        if (i >= 0 && j >= 0 && i < Nx && j < Ny) {
            if (mask)derivative_no_pressure(fluid, dot_cx, dot_cy, time_t, i, j);
            else set_boundaries(fluid, time_t, i, j);
            old_cx[j] = fluid->speed->cx[index];
            old_cy[j] = fluid->speed->cy[index];

            shared_cx[j] = old_cx[j] + dt_6 * dot_cx[j];
            shared_cy[j] = old_cy[j] + dt_6 * dot_cy[j];
        }
        g.sync();

        if (mask) {
            fluid->speed->cx[index] += hdt * dot_cx[j];
            fluid->speed->cy[index] += hdt * dot_cy[j];
        }
        g.sync();

        if (i >= 0 && j >= 0 && i < Nx && j < Ny) {
            if (mask)derivative_no_pressure(fluid, dot_cx, dot_cy, time_t, i, j);
            else set_boundaries(fluid, time_t, i, j);
            shared_cx[j] += dt_3 * dot_cx[j];
            shared_cy[j] += dt_3 * dot_cy[j];
        }

        g.sync();

        if (mask) {
            fluid->speed->cx[index] += old_cx[j] + hdt * dot_cx[j];
            fluid->speed->cy[index] += old_cy[j] + hdt * dot_cy[j];
        }

        g.sync();

        if (i >= 0 && j >= 0 && i < Nx && j < Ny) {
            if (mask)derivative_no_pressure(fluid, dot_cx, dot_cy, time_t, i, j);
            else set_boundaries(fluid, time_t, i, j);
            shared_cx[j] += dt_3 * dot_cx[j];
            shared_cy[j] += dt_3 * dot_cy[j];
        }
        g.sync();

        if (mask) {

            fluid->speed->cx[index] += old_cx[j] + dt * dot_cx[j];
            fluid->speed->cy[index] += old_cy[j] + dt * dot_cy[j];
        }
        g.sync();

        if (i >= 0 && j >= 0 && i < Nx && j < Ny) {
            if (mask)derivative_no_pressure(fluid, dot_cx, dot_cy, time_t, i, j);
            else set_boundaries(fluid, time_t, i, j);
        }
        g.sync();

        if (i >= ghost_zone && j >= ghost_zone && i < Nx - ghost_zone && j < Ny - ghost_zone){
            if (mask) {
                fluid->speed->cx[index] = shared_cx[j] + dt_6 * dot_cx[j];
                fluid->speed->cy[index] = shared_cy[j] + dt_6 * dot_cy[j];

            }
            fluid->pressure->v[index] = 0;
        }
    }
}

void set_boundaries_CPU(struct Fluid* fluid,double time_t){
    const int Nx=fluid->speed->Nx;
    const int Ny=fluid->speed->Ny;
    const double dx=fluid->speed->dx;
    const double dy=fluid->speed->dy;
    const int ghost_zone=fluid->speed->ghost_zone;
    for (int i = 0; i < Nx; ++i) {
        //y=0 boundary
        fluid->speed->cx[i]=0;
        fluid->speed->cy[i]=0;
        //y=1 boundary
        fluid->speed->cx[i+Nx*(Ny-1)]=0;
        fluid->speed->cy[i+Nx*(Ny-1)]=0;
    }
    for (int j = 0; j < Ny; ++j) {
        //x=0 boundary
        fluid->speed->cx[Nx*j]=USUAL_VX_INPUT;
        if(j>55)fluid->speed->cx[Nx*j]=-0;
        fluid->speed->cy[Nx*j]=0;
        //x=1 boundary
        fluid->speed->cx[Nx*j+Nx-1]=0;
        if(j>55)fluid->speed->cx[Nx*j+Nx-1]=-USUAL_VX_OUTPUT;
        fluid->speed->cy[Nx*j+Nx-1]=0;
        fluid->smoke->v[Nx*j]=0;
        fluid->smoke->v[Nx*j+Nx-1]=0;
        if(j>30 && j<40){
            fluid->smoke->v[Nx*j]=1;

        }
        if(j>70 && j<80){
            fluid->smoke->v[Nx*j+Nx-1]=1;
        }
        /*if(j>40+20* cos(0.25*time_t) && j<60+20* cos(0.25*time_t)){
            fluid->speed->cx[Nx*j]=12;
            fluid->smoke->v[Nx*j]=1;
            //printf("%f %f\n",40+20* cos(0.5*time_t),time_t);
        }*/
    }
    for (int j = ghost_zone; j < Ny-ghost_zone; ++j) {
        for (int i = ghost_zone; i < Nx - ghost_zone; ++i) {
            int index = INDEX(i, j, Nx);
            if(fluid->mask[index]>0)continue;
            fluid->speed->cx[index]=0;
            fluid->speed->cy[index]=0;
            fluid->smoke->v[index]=-1;
        }
    }

}

void solve_pressure2(struct Fluid* fluid,double time_t,double dt){
    const int Nx=fluid->speed->Nx;
    const int Ny=fluid->speed->Ny;
    const double dx=fluid->speed->dx;
    const double dy=fluid->speed->dy;
    const int ghost_zone=fluid->speed->ghost_zone;
    divergence(fluid->speed,fluid->tempScalar);

    for (int ite = 0; ite < 10000; ++ite) {
        double error=0;
        double maxe=0;

        divergence(fluid->speed,fluid->tempScalar);
        for (int j = ghost_zone; j < Ny-ghost_zone; ++j) {
            for (int i = ghost_zone; i < Nx-ghost_zone; ++i) {
                int index=INDEX(i,j,Nx);
                if(fluid->mask[index]==0)continue;
                double value_e=fluid->tempScalar->v[index];
                error+=value_e*value_e;
                maxe= fmax(fabs(value_e),maxe);
            }
        }
        error/=(Nx-2)*(Ny-2);
        if(error<1e-4 ||ite>=1000){
            printf("Iteration: %d;Error:%20.16e Max:%20.16e\n",ite,error,maxe);
            break;
        }

        if(ite==0){
            printf("Initialisation Error:%20.16e Max:%20.16e\n",error,maxe);
        }

        for (int k = 0; k < 100; ++k) {
            for (int j = ghost_zone; j < Ny-ghost_zone; ++j) {
                for (int i = ghost_zone; i < Nx-ghost_zone; ++i) {
                    int index=INDEX(i,j,Nx);
                    if(fluid->mask[index]==0)continue;
                    double div=2*dx*FD_2O_CEN(fluid->speed->cx,INDEX(i,j,Nx),1,dx)+2*dy*FD_2O_CEN(fluid->speed->cy,INDEX(i,j,Nx),Nx,dy);
                    int count_neighs=4;
                    if(fluid->mask[INDEX(i-1,j,Nx)]==0)count_neighs--;
                    if(fluid->mask[INDEX(i+1,j,Nx)]==0)count_neighs--;
                    if(fluid->mask[INDEX(i,j-1,Nx)]==0)count_neighs--;
                    if(fluid->mask[INDEX(i,j+1,Nx)]==0)count_neighs--;

                    fluid->speed->cy[INDEX(i,j-1,Nx)]+=div/count_neighs*1.5;
                    fluid->speed->cy[INDEX(i,j+1,Nx)]-=div/count_neighs*1.5;
                    fluid->speed->cx[INDEX(i-1,j,Nx)]+=div/count_neighs*1.5;
                    fluid->speed->cx[INDEX(i+1,j,Nx)]-=div/count_neighs*1.5;
                    fluid->pressure->v[index]+=div/4*fluid->density*dx/dt;
                }
            }
            set_boundaries_CPU(fluid,time_t);
        }
    }

}

__global__
void printer(struct Fluid* fluid,double time_t,double dt) {
    printf("Enter\n");
}

__global__
void advect(struct Fluid* fluid,double time_t,double delta_t){
    cooperative_groups::grid_group g = cooperative_groups::this_grid();
    delta_t= fluid->dt;
    time_t=fluid->t;
    __shared__ double shared_cx[NY_VALUE];
    __shared__ double shared_cy[NY_VALUE];
    __shared__ double shared_v[NY_VALUE];
    int j =  threadIdx.x;

    const int Nx=fluid->speed->Nx;
    const int Ny=fluid->speed->Ny;
    const double dx=fluid->speed->dx;
    const double dy=fluid->speed->dy;
    const int ghost_zone=fluid->speed->ghost_zone;
    int rep=(Nx+gridDim.x-1)/gridDim.x;
    for(int id=0;id<rep;id++) {
        int i=blockIdx.x+id*gridDim.x;
        int index = INDEX_GPU(i, j, Nx);

        if (i >= ghost_zone && j >= ghost_zone && i < Nx - ghost_zone && j < Ny - ghost_zone)
            if (fluid->mask[index] != 0) {

                double newX = i - delta_t / dx * fluid->speed->cx[index];

                double newY = j - delta_t / dy * fluid->speed->cy[index];
                newX = fmax(0.0, fmin(0.0 + Nx - 1, newX));
                newY = fmax(0.0, fmin(0.0 + Ny - 1, newY));
                int nx = (int) newX;
                int ny = (int) newY;

                double difx = newX - nx;
                double dify = newY - ny;
                shared_cx[j] = (1 - difx) * (1 - dify) * fluid->speed->cx[INDEX_GPU(nx, ny, Nx)]
                               + (difx) * (1 - dify) * fluid->speed->cx[INDEX_GPU(nx + 1, ny, Nx)]
                               + (1 - difx) * (dify) * fluid->speed->cx[INDEX_GPU(nx, ny + 1, Nx)]
                               + (difx) * (dify) * fluid->speed->cx[INDEX_GPU(nx + 1, ny + 1, Nx)];
                shared_cy[j] = (1 - difx) * (1 - dify) * fluid->speed->cy[INDEX_GPU(nx, ny, Nx)]
                               + (difx) * (1 - dify) * fluid->speed->cy[INDEX_GPU(nx + 1, ny, Nx)]
                               + (1 - difx) * (dify) * fluid->speed->cy[INDEX_GPU(nx, ny + 1, Nx)]
                               + (difx) * (dify) * fluid->speed->cy[INDEX_GPU(nx + 1, ny + 1, Nx)];
                shared_v[j] =
                        (1 - difx) * (1 - dify) * fmax(fluid->smoke->v[INDEX_GPU(nx, ny, Nx)], 0.0)
                        + (difx) * (1 - dify) * fmax(fluid->smoke->v[INDEX_GPU(nx + 1, ny, Nx)], 0.0)
                        + (1 - difx) * (dify) * fmax(fluid->smoke->v[INDEX_GPU(nx, ny + 1, Nx)], 0.0)
                        + (difx) * (dify) * fmax(fluid->smoke->v[INDEX_GPU(nx + 1, ny + 1, Nx)], 0.0);
            }

        g.sync();

        if (i >= ghost_zone && j >= ghost_zone && i < Nx - ghost_zone && j < Ny - ghost_zone)
            if (fluid->mask[index] != 0) {
                fluid->speed->cx[index] = shared_cx[j];
                fluid->speed->cy[index] = shared_cy[j];
                fluid->smoke->v[index] = fluid->smoke->v[index] * 0.1 + 0.9 * shared_v[j];
            }
    }
}


int main(int argc, char **argv) {
    const int Nx=NX_VALUE;
    const int Ny=NY_VALUE;
    const int ghost_zone=1;
    const double dx=70.0/(Nx-1);
    const double dy=70.0/(Ny-1);

    double* dtp;
    CUDA_CHECK(hipMallocManaged((void**)&dtp, sizeof(double )));
    *dtp=0.1;
    printf("DT:%f\n",*dtp);
    double dt=*dtp;
    struct Fluid* fluid;
    CUDA_CHECK(hipMallocManaged(&fluid, sizeof(struct Fluid)));

    printf("Entraine\n");
    initFluid(fluid,Nx,Ny,ghost_zone,dx,dy);
    printf("Fin init\n");

    char* centered_mask;
    CUDA_CHECK(hipMallocManaged(&(centered_mask), Nx * Ny * sizeof(char)));
    FILE *fichier;
    // Chemin du fichier à ouvrir en mode lecture ("r" pour read)
    const char *chemin_fichier = "simMask/duck110.txt";
    fichier = fopen(chemin_fichier, "r");
    // Vérifier si l'ouverture du fichier a réussi
    if (fichier == NULL) {
        fprintf(stderr, "Erreur lors de l'ouverture du fichier.\n");
        return 1; // Quitter le programme avec code d'erreur
    }

    for(int i =0;i<Nx;i++){
        for(int j =0;j<Ny;j++){
            fluid->mask[INDEX(i,j,Nx)]=1;
            centered_mask[INDEX(i,j,Nx)]=1;
        }
    }

    double value=0;
    int count=0;
    while (fscanf(fichier, "%lf", &value) == 1) {
        if(count>=Nx*Ny)break;
        int i=count/Nx;
        int j=count%Nx;

        centered_mask[count]=1;
        fluid->mask[count]=1;
       /* if(value<0.5 ){//&& i>5 && i<Nx-5 && j>5 && j<Ny-5){
            centered_mask[count]=0;
            fluid->mask[count]=0;
            fluid->smoke->v[count]=-1;
        }*/
        count++;
    }
    printf("Fin %d/%d\n",count,Nx*Ny);
    fclose(fichier);

    fluid->density=1;
    fluid->viscosity=0.001;



//------------------------------------------------------------------------

//------------------------------------------------------------------------

    int numBlocksPerSm = 0;
    // Number of threads my_kernel will be launched with
    int numThreads = Ny;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksPerSm, (void*) RK4, numThreads, 0);
    // launch
    double* time_p;
    CUDA_CHECK(hipMallocManaged((void**)&time_p, sizeof(double )));
    *time_p=0;
    // Lancement du kernel avec hipLaunchCooperativeKernel
    dim3 dimBlock(numThreads, 1, 1);
    dim3 dimGrid(deviceProp.multiProcessorCount*numBlocksPerSm, 1, 1);
    if (dimGrid.x > Nx)
    {
        dimGrid.x = Nx;
    }
    printf("numBlocks = %d, blockSize = %d, totalThreads=%d %d\n", dimGrid.x, dimBlock.x, dimGrid.x * dimBlock.x,deviceProp.multiProcessorCount);

    double  begin=get_time();
    output_gf(fluid);

    int iterations=0;

    while(*time_p<=100){
        fluid->dt=dt;
        fluid->t=*time_p;
        struct {
            struct Fluid* fluid;
            double t;
            double dt;
        } params = { fluid,*time_p,*dtp };


        void* kernelArgs[] = { (void*)&params };

        printf("Time:%f\n",*time_p);
        //dont remove necessary but idk why
        if(*time_p==0) {
            printer<<<1, 1>>>(fluid, *time_p, 0);
            CUDA_CHECK(hipDeviceSynchronize());
        }

        CUDA_CHECK(hipLaunchCooperativeKernel((void*) RK4, dimGrid, dimBlock, kernelArgs));
        CUDA_CHECK(hipDeviceSynchronize());

        solve_pressure2(fluid,*time_p,dt);

       hipLaunchCooperativeKernel((void*) advect, dimGrid, dimBlock, kernelArgs);
        CUDA_CHECK(hipDeviceSynchronize());
        *time_p+=dt;
        iterations+=1;
        if(iterations%(10)==0){
            output_gf(fluid);
        }
    }

    printf("Fin time:%f Iteration:%d\n",get_time()-begin,iterations);

    freeFieldScalar(fluid->pressure);
    freeFieldScalar(fluid->tempScalar);
    freeField3D(fluid->speed);
    freeField3D(fluid->temp2D);
    freeField3D(fluid->old);
    freeFieldScalar(fluid->smoke);
    hipFree(centered_mask);
    hipFree(fluid->speed);
    hipFree(fluid->temp2D);
    hipFree(fluid->tempScalar);
    hipFree(fluid->pressure);
    hipFree(fluid->old);
    hipFree(fluid->smoke);
    hipFree(fluid->mask);
    return 0;
}

/*
        double error=99999;
        int ite=0;
        while(error>1e-5 && ite<100){
            ite++;
            error=0;

            hipLaunchCooperativeKernel((void*) solve_pressure, dimGrid, dimBlock, kernelArgs);
            CUDA_CHECK(hipDeviceSynchronize());
            double maxi=0;
            int im=0;int jm=0;
            for(int i =ghost_zone;i<Nx-ghost_zone;i++){
                for(int j =ghost_zone;j<Ny-ghost_zone;j++){
                    error+=fluid->tempScalar->v[INDEX(i,j,Nx)];
                    if(fluid->tempScalar->v[INDEX(i,j,Nx)]>maxi){
                        im=i;jm=j;
                        maxi=fluid->tempScalar->v[INDEX(i,j,Nx)];
                    }
                }
            }
            //printf("%f\n",fluid->speed->cx[INDEX(8,7,Nx)]);
            error/=(Nx-2)*(Ny-2);
            printf("Iteration: %d;Error:%20.16e %d %d %d %f\n",ite,error,im,jm, INDEX(im,jm,Nx),maxi);
        }
        printf("Iteration: %d;Error:%20.16e\n",ite,error);*/

void initFluid(struct Fluid *fluid, int Nx, int Ny,int ghost_zone,double dx,double dy) {

    // Allouer de la mémoire pour chaque structure et tableau sur le GPU
    CUDA_CHECK(hipMallocManaged(&(fluid->speed), sizeof(struct Field2D)));
    CUDA_CHECK(hipMallocManaged(&(fluid->temp2D), sizeof(struct Field2D)));
    CUDA_CHECK(hipMallocManaged(&(fluid->old), sizeof(struct Field2D)));
    CUDA_CHECK(hipMallocManaged(&(fluid->pressure), sizeof(struct FieldScalar)));
    CUDA_CHECK(hipMallocManaged(&(fluid->tempScalar), sizeof(struct FieldScalar)));
    CUDA_CHECK(hipMallocManaged(&(fluid->smoke), sizeof(struct FieldScalar)));
    CUDA_CHECK(hipMallocManaged(&(fluid->mask), Nx * Ny * sizeof(char)));

    if (fluid->speed == NULL || fluid->temp2D == NULL ||
        fluid->old == NULL || fluid->pressure == NULL || fluid->tempScalar == NULL ||
        fluid->smoke == NULL || fluid->mask == NULL) {
        printf("Error dynamic allocation for fluid\n");
        exit(-1);
    }
    // Initialiser d'autres champs si nécessaire
    initField2D(fluid->speed,Nx,Ny,ghost_zone,dx,dy);
    initField2D(fluid->temp2D,Nx,Ny,ghost_zone,dx,dy);
    initField2D(fluid->old,Nx,Ny,ghost_zone,dx,dy);
    initFieldScalar(fluid->pressure,Nx,Ny,ghost_zone,dx,dy);
    initFieldScalar(fluid->tempScalar,Nx,Ny,ghost_zone,dx,dy);
    initFieldScalar(fluid->smoke,Nx,Ny,ghost_zone,dx,dy);
}

void output_gf(struct Fluid* fluid)
{

    static int counter = 0;
    const int Nx=fluid->speed->Nx;
    const int Ny=fluid->speed->Ny;
    const double dx=fluid->speed->dx;
    const double dy=fluid->speed->dy;

    char name_buff[1024];
    snprintf(name_buff, 1024, "../results/%07d.asc", counter);
    FILE *ofile = fopen(name_buff, "w");
    for (int j = 0; j < Ny; ++j)
    {
        for (int i = 0; i < Nx; ++i)
        {
            const int ij = INDEX(i, j, Nx);
            fprintf(ofile, "%20.16e %20.16e %20.16e %20.16e %20.16e %20.16e\n", i * dx,j * dy,
                    fluid->pressure->v[ij],fluid->speed->cx[ij],fluid->speed->cy[ij],fluid->smoke->v[ij]);
        }
    }
    ++counter;

    fclose(ofile);
    ofile = NULL;
}

double get_time(void)
{
    struct timeval tv;
    gettimeofday(&tv, NULL);

    return (tv.tv_sec) + 1.0e-6 * tv.tv_usec;
}

__global__
void solve_pressure(struct Fluid* fluid,double time_t,double dt){
    dt= fluid->dt;
    time_t=fluid->t;
    cooperative_groups::grid_group g = cooperative_groups::this_grid();

    int j =  threadIdx.x;
    const int Nx=fluid->speed->Nx;

    const int Ny=fluid->speed->Ny;
    const double dx=fluid->speed->dx;
    const double dy=fluid->speed->dy;
    const int ghost_zone=fluid->speed->ghost_zone;

    int rep=(Nx+gridDim.x-1)/gridDim.x;

    for (int k = 0; k < 250; ++k) {
        for(int id=0;id<rep;id++) {
            int i = blockIdx.x + id * gridDim.x;
            int index = INDEX_GPU(i, j, Nx);
            int mask = i >= ghost_zone && j >= ghost_zone && i < Nx - ghost_zone &&
                       j < Ny - ghost_zone;
            if(mask){
                mask=fluid->mask[index] != 0;
            }
            int count_neighs = 4;
            if (mask) {
                if (fluid->mask[INDEX_GPU(i - 1, j, Nx)] == 0 || i - 1 <= ghost_zone)count_neighs--;
                if (fluid->mask[INDEX_GPU(i + 1, j, Nx)] == 0 || i + 1 > Nx - ghost_zone)count_neighs--;
                if (fluid->mask[INDEX_GPU(i, j - 1, Nx)] == 0 || j - 1 <= ghost_zone)count_neighs--;
                if (fluid->mask[INDEX_GPU(i, j + 1, Nx)] == 0 || j + 1 > Ny - ghost_zone)count_neighs--;
            }
            double div = 0;

            if (mask) {
                div = 2 * dx * FD_2O_CEN_GPU(fluid->speed->cx, index, 1, dx) +
                      2 * dy * FD_2O_CEN_GPU(fluid->speed->cy, index, Nx, dy);
                fluid->pressure->v[index] = 0.5 * div;
                fluid->speed->cx[index] += 0.5 * (fluid->pressure->v[INDEX_GPU(i + 1, j, Nx)] -
                                                  fluid->pressure->v[INDEX_GPU(i - 1, j, Nx)]);
                fluid->speed->cy[index] += 0.5 * (fluid->pressure->v[INDEX_GPU(i, j + 1, Nx)] -
                                                  fluid->pressure->v[INDEX_GPU(i, j - 1, Nx)]);
                fluid->pressure->v[index] += div / 4 * fluid->density * dx / dt;
            } else {
                fluid->pressure->v[index] = 0;
                if (fluid->mask[index] != 0)set_boundaries(fluid, time_t, i, j);
            }
            g.sync();
        }
    }

    for(int id=0;id<rep;id++) {
        int i = blockIdx.x + id * gridDim.x;
        int index = INDEX_GPU(i, j, Nx);
        int mask = i >= ghost_zone && j >= ghost_zone && i < Nx - ghost_zone &&
                   j < Ny - ghost_zone;
        if(mask){
            mask=fluid->mask[index] != 0;
        }
        if (mask) {
            double div = FD_2O_CEN_GPU(fluid->speed->cx, index, 1, dx) + FD_2O_CEN_GPU(fluid->speed->cy, index, Nx, dy);
            fluid->tempScalar->v[index] = div * div;

        } else {
            if (fluid->mask[index] != 0)set_boundaries(fluid, time_t, i, j);
        }
    }
}



