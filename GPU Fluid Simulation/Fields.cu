//
// Created by bapti on 19/04/2024.
//
#include <stdio.h>
#include <stdlib.h>
#include "Fields.h"
#include "derivative.h"



void laplacien(struct Field2D* vectors,struct FieldScalar* result){
    for (int j = vectors->ghost_zone; j < vectors->Ny-vectors->ghost_zone; ++j) {
        for (int i = vectors->ghost_zone; i < vectors->Nx-vectors->ghost_zone; ++i) {
            result->v[INDEX(i,j,vectors->Nx)]= SD_2O_CEN(vectors->cx,INDEX(i,j,vectors->Nx),1,vectors->dx)+
                    SD_2O_CEN(vectors->cy,INDEX(i,j,vectors->Nx),vectors->Nx,vectors->dy);
        }
    }
}

void gradient(double* value,struct Field2D* result){
    for (int j = result->ghost_zone; j < result->Ny-result->ghost_zone; ++j) {
        for (int i = result->ghost_zone; i < result->Nx-result->ghost_zone; ++i) {
            result->cx[INDEX(i,j,result->Nx)]= FD_2O_CEN(value,INDEX(i,j,result->Nx),1,result->dx);
            result->cy[INDEX(i,j,result->Nx)]= FD_2O_CEN(value,INDEX(i,j,result->Nx),result->Nx,result->dy);
        }
    }


    
}

void divergence(struct Field2D* vectors,struct FieldScalar* result){
    //interior
    for (int j = vectors->ghost_zone; j < vectors->Ny-vectors->ghost_zone; ++j){
        for (int i = vectors->ghost_zone; i < vectors->Nx-vectors->ghost_zone; ++i) {
            result->v[INDEX(i,j,vectors->Nx)]= FD_2O_CEN(vectors->cx,INDEX(i,j,vectors->Nx),1,vectors->dx)+FD_2O_CEN(vectors->cy,INDEX(i,j,vectors->Nx),result->Nx,vectors->dy);
        }
    }
}

void initField2D(struct Field2D *field2D, int Nx, int Ny, int ghost, double dx, double dy) {
    // Allouer de la mémoire sur le GPU pour cx et cy
    CUDA_CHECK(hipMallocManaged(&(field2D->cx), Nx * Ny * sizeof(double)));

    CUDA_CHECK(hipMallocManaged(&(field2D->cy), Nx * Ny * sizeof(double)));

    if (field2D->cx == NULL || field2D->cy == NULL) {
        printf("Error dynamic allocation 2D field\n");
        exit(-1);
    }
    // Initialiser la mémoire allouée à zéro
    CUDA_CHECK(hipMemset(field2D->cx, 0, Nx * Ny * sizeof(double)));
    CUDA_CHECK(hipMemset(field2D->cy, 0, Nx * Ny * sizeof(double)));

    // Initialiser les autres champs de la structure
    field2D->Nx = Nx;
    field2D->Ny = Ny;
    field2D->dx = dx;
    field2D->dy = dy;
    field2D->ghost_zone = ghost;
}


void initFieldScalar(struct FieldScalar* fieldScalar,int Nx,int Ny,int ghost, double dx,double dy){
    fieldScalar->Nx=Nx;
    fieldScalar->Ny=Ny;
    fieldScalar->dx=dx;
    fieldScalar->dy=dy;
    fieldScalar->ghost_zone=ghost;
    int size=Nx*Ny;
    CUDA_CHECK(hipMallocManaged(&(fieldScalar->v), Nx * Ny * sizeof(double)));
    if(fieldScalar->v==NULL){
        printf("Error dynamic allocation scalar field\n");
        exit(-1);
    }
    CUDA_CHECK(hipMemset(fieldScalar->v, 0, Nx * Ny * sizeof(double)));
}
void freeField3D(struct Field2D* field2D){
    hipFree(field2D->cx);
    hipFree(field2D->cy);
}
void freeFieldScalar(struct FieldScalar* fieldScalar){
    hipFree(fieldScalar->v);
}